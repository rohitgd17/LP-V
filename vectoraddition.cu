#include "hip/hip_runtime.h"
%%writefile cuda_example.cu
#include <bits/stdc++.h>
using namespace std;

__global__ void add(int *x, int *y, int *z)
{
    int id = blockIdx.x;
    z[id] = x[id] + y[id];
}

int main()
{
    int a[6]={1,2,3,4,5,6}, b[6]={1,2,3,4,5,6}, c[6], *d,*e,*f;

    int n=6;
   


    hipMalloc((void**)&d,6*sizeof(int));
    hipMalloc((void**)&e,6*sizeof(int));
    hipMalloc((void**)&f,6*sizeof(int));

    hipMemcpy(d,a,6*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(e,b,6*sizeof(int),hipMemcpyHostToDevice);

    add<<<6,1>>>(d,e,f);

    hipMemcpy(c,f,6*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        cout<<c[i]<<" ";
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);


    return 0;

}

// % % writefile cuda_example.cu
// #include <iostream>
//     using namespace std;

// __global__ void add(int *A, int *B, int *C, int size)
// {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;

//     if (tid < size)
//     {
//         C[tid] = A[tid] + B[tid];
//     }
// }

// void initialize(int *vector, int size)
// {
//     for (int i = 0; i < size; i++)
//     {
//         vector[i] = rand() % 10;
//     }
// }

// void print(int *vector, int size)
// {
//     for (int i = 0; i < size; i++)
//     {
//         cout << vector[i] << " ";
//     }
//     cout << endl;
// }

// int main()
// {
//     int N = 4;
//     int *A, *B, *C;

//     int vectorSize = N;
//     size_t vectorBytes = vectorSize * sizeof(int);

//     A = new int[vectorSize];
//     B = new int[vectorSize];
//     C = new int[vectorSize];

//     initialize(A, vectorSize);
//     initialize(B, vectorSize);

//     cout << "Vector A: ";
//     print(A, N);
//     cout << "Vector B: ";
//     print(B, N);

//     int *X, *Y, *Z;
//     hipMalloc(&X, vectorBytes);
//     hipMalloc(&Y, vectorBytes);
//     hipMalloc(&Z, vectorBytes);

//     hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
//     hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

//     int threadsPerBlock = 256;
//     int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

//     add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

//     hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

//     cout << "Addition: ";
//     print(C, N);

//     delete[] A;
//     delete[] B;
//     delete[] C;

//     hipFree(X);
//     hipFree(Y);
//     hipFree(Z);

//     return 0;
// }